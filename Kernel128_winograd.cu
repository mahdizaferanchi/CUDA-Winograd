#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <assert.h>
#include <xmmintrin.h>
#include <immintrin.h>

#include "hipDNN.h"
#include "util.h"
#include "Kernel128_winograd.h"


#define cudaCheckError() {																\
	hipError_t e=hipGetLastError();													\
	if(e!=hipSuccess) {																\
		printf("Cuda failure %s:%d:'%s'\n",__FILE__,__LINE__,hipGetErrorString(e));	\
		exit(EXIT_FAILURE);																\
	}																					\
}

#define MY_KERNEL 1

#define d(input, i, j, Inz) ( input[Inz + i*768 + (j<<7)] )
#define relu(x) (x > 0 ? x : 0)

__global__ void kernel_128_winograd_BtdB(float *pInputs, float *pOutputs) {
	int Inx = blockIdx.x<<2, Iny0 = blockIdx.y<<2, Iny1 = threadIdx.y, Inz = threadIdx.x;
	int Iny = Iny0+Iny1, stride_r = 2048, stride_c = 128; // 2048 = 16*128
	int c_glb_start = Inx*stride_r + Iny*stride_c + Inz, c_input = Iny1*stride_c + Inz;

	extern __shared__ float input[];

	int tmp[6] = {0, 768, 1536, 2304, 3072, 3840}; // 768 = 6*128
	for (int i = 0; i < 6; i++) {
		input[c_input + tmp[i]] = pInputs[c_glb_start + i*stride_r];
	}
	__syncthreads();

	float BTd[6];
	switch(Iny1) {
		case 0:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 0, j, Inz)*4 - d(input, 2, j, Inz)*5 + d(input, 4, j, Inz);
			}
			break;
		case 1:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 + d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 2:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 2, j, Inz)*4 - d(input, 3, j, Inz) + d(input, 4, j, Inz);
			}
			break;
		case 3:
			for (int j = 0; j < 6; j++) {
				BTd[j] = -d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) + d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 4:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*2 - d(input, 2, j, Inz) - d(input, 3, j, Inz)*2 + d(input, 4, j, Inz);
			}
			break;
		case 5:
			for (int j = 0; j < 6; j++) {
				BTd[j] = d(input, 1, j, Inz)*4 - d(input, 3, j, Inz)*5 + d(input, 5, j, Inz);
			}
			break;
	}
	__syncthreads();

	int tmp_offset = Iny1*768+Inz;
	for (int i = 0; i < 6; i++) {
		input[tmp_offset + i*stride_c] = BTd[i];
	}
	__syncthreads();

	float BTdB[6];
	switch(Iny1) {
		case 0:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 0, Inz) - 5*d(input, i, 2, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 1:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) + d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 2:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 4*d(input, i, 2, Inz) - d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 3:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = -2*d(input, i, 1, Inz) - d(input, i, 2, Inz) + 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 4:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 2*d(input, i, 1, Inz) - d(input, i, 2, Inz) - 2*d(input, i, 3, Inz) + d(input, i, 4, Inz);
			}
			break;
		case 5:
			for (int i = 0; i < 6; i++) {
				BTdB[i] = 4*d(input, i, 1, Inz) - 5*d(input, i, 3, Inz) + d(input, i, 5, Inz);
			}
			break;
	}
	__syncthreads();

	for (int i = 0; i < 6; i++) {
		pOutputs[(Iny1 + i*6)*2048 + (blockIdx.x*4+blockIdx.y)*128 + Inz] = BTdB[i];
	}
}

__global__ void kernel_128_winograd_AtIA_v2(float *pInputs, float *pBiases, float *pScales, float *pOutputs) { 
	int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = threadIdx.x;

	// int c_input = Inx*6 + Iny;

	 float bias, scale;
	 extern __shared__ float input[];

	for (int i = 0; i < 6; ++i) {
		input[(i*6 + Iny) + kz*36] = pInputs[(i*6 + Iny)*16*128 + (Tilex*4+Tiley)*128 + kz];
	}
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp[6];
	for (int i = 0; i < 4; ++i) {
		switch(i) {
			case 0:
				tmp[i] = input[kz*36 + Iny] + input[kz*36 + 6+Iny] + input[kz*36 + 12+Iny] + input[kz*36 + 18+Iny] + input[kz*36 + 24+Iny];
				break;
			case 1:
				tmp[i] = input[kz*36 + 6+Iny] - input[kz*36 + 12+Iny] + 2*input[kz*36 + 18+Iny] - 2*input[kz*36 + 24+Iny];
				break;
			case 2:
				tmp[i] = input[kz*36 + 6+Iny] + input[kz*36 + 12+Iny] + 4*input[kz*36 + 18+Iny] + 4*input[kz*36 + 24+Iny];
				break;
			case 3:
				tmp[i] = input[kz*36 + 6+Iny] - input[kz*36 + 12+Iny] + 8*input[kz*36 + 18+Iny] - 8*input[kz*36 + 24+Iny] + input[kz*36 + 30+Iny];
				break;
		}
	}
	
	__syncthreads();

	for (int i = 0; i < 4; i++) {
		input[kz*36 + i*6 + Iny] = tmp[i];
	}
	__syncthreads();


	int x;
	float o;
	switch(Iny) {
		case 0:
			for (int i = 0; i < 4; ++i){
				if ((Tilex == 3 && i > 1)) return;
				x = i*6;
				o = scale*(input[kz*36 + x]+input[kz*36 + x+1]+input[kz*36 + x+2]+input[kz*36 + x+3]+input[kz*36 + x+4])+ bias;
				pOutputs[(((Tilex<<2)+1+i)*16 + (Tiley<<2)+1)*128 + kz] = o > 0 ? o : 0;
			}
			break;
		case 1:
			for (int i = 0; i < 4; ++i){
				if ((Tilex == 3 && i > 1)) return;
				x = i*6;
				o = scale*(input[kz*36 + x+1] - input[kz*36 + x+2] + 2*input[kz*36 + x+3] - 2*input[kz*36 + x+4]) + bias;
				pOutputs[(((Tilex<<2)+1+i)*16 + (Tiley<<2)+2)*128 + kz] = o > 0 ? o : 0;
			}
			break;
		case 2:
			if (Tiley == 3) break;
			for (int i = 0; i < 4; ++i){
				if ((Tilex == 3 && i > 1)) return;
				x = i*6;
				o = scale*(input[kz*36 + x+1] + input[kz*36 + x+2] + 4*input[kz*36 + x+3] + 4*input[kz*36 + x+4]) + bias;
				pOutputs[(((Tilex<<2)+1+i)*16 + (Tiley<<2)+3)*128 + kz] = o > 0 ? o : 0;
			}
			break;
		case 3:
			if (Tiley == 3) break;
			for (int i = 0; i < 4; ++i){
				if ((Tilex == 3 && i > 1)) return;
				x = i*6;
				o = scale*(input[kz*36 + x+1] - input[kz*36 + x+2] + 8*input[kz*36 + x+3] - 8*input[kz*36 + x+4] + input[kz*36 + x+5]) + bias;
				pOutputs[(((Tilex<<2)+1+i)*16 + (Tiley<<2)+4)*128 + kz] = o > 0 ? o : 0;
			}
			break;
	}
}

__global__ void kernel_128_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) { 
	int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
	// int Tilex = threadIdx.x, Tiley = threadIdx.y, Iny = blockIdx.y, kz = blockIdx.z, Inx = blockIdx.x;

	int c_input = Inx*6 + Iny;

	__shared__ float bias, scale;
	 extern __shared__ float input[];

	input[c_input] = pInputs[c_input*16*128 + (Tilex*4+Tiley)*128 + kz];
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp = 0;
	switch(Inx) {
		case 0:
			tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
			break;
		case 1:
			tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
			break;
		case 2:
			tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
			break;
		case 3:
			tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
			break;
	}
	__syncthreads();

	input[c_input] = tmp;
	__syncthreads();

	if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;

	int x;
	float o;
	switch(Iny) {
		case 0:
			x = Inx*6;
			o = scale*(input[x]+input[x+1]+input[x+2]+input[x+3]+input[x+4])+ bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+1)*128 + kz] = o > 0 ? o : 0;
			break;
		case 1:
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 2*input[x+3] - 2*input[x+4]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+2)*128 + kz] = o > 0 ? o : 0;
			break;
		case 2:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] + input[x+2] + 4*input[x+3] + 4*input[x+4]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+3)*128 + kz] = o > 0 ? o : 0;
			break;
		case 3:
			if (Tiley == 3) break;
			x = Inx*6;
			o = scale*(input[x+1] - input[x+2] + 8*input[x+3] - 8*input[x+4] + input[x+5]) + bias;
			pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+4)*128 + kz] = o > 0 ? o : 0;
			break;
	}
}

__global__ void kernel_128_winograd_AtIA_avgp(float *pInputs, float *pBiases, float *pScales, float *pOutputs) { 
	int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
	// int Tilex = threadIdx.x, Tiley = threadIdx.y, Iny = blockIdx.y, kz = blockIdx.z, Inx = blockIdx.x;

	int c_input = Inx*6 + Iny;

	__shared__ float bias, scale;
	 extern __shared__ float input[];

	input[c_input] = pInputs[c_input*16*128 + (Tilex*4+Tiley)*128 + kz];
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp = 0;
	switch(Inx) {
		case 0:
			tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
			break;
		case 1:
			tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
			break;
		case 2:
			tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
			break;
		case 3:
			tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
			break;
	}
	__syncthreads();

	input[c_input] = tmp;
	__syncthreads();

	if (Inx >= 3 || Inx == 1 || (Tilex == 3 && Inx > 1)) return;
	int x0, x1;
	float o, o00, o01, o10, o11;
	switch(Iny) {
		case 0:
			x0 = Inx*6, x1 = (Inx+1)*6;
			o00 = scale*(input[x0]+input[x0+1]+input[x0+2]+input[x0+3]+input[x0+4])+bias;
			o01 = scale*(input[x0+1] - input[x0+2] + 2*input[x0+3] - 2*input[x0+4])+bias;
			o10 = scale*(input[x1]+input[x1+1]+input[x1+2]+input[x1+3]+input[x1+4])+bias;
			o11 = scale*(input[x1+1] - input[x1+2] + 2*input[x1+3] - 2*input[x1+4])+bias;
			o = (0.25)*(relu(o00)+relu(o01)+relu(o10)+relu(o11));
			// o = fmaxf(fmaxf(relu(o00), relu(o01)), fmaxf(relu(o10), relu(o11)));
			pOutputs[(((Tilex<<1)+1+Inx/2)*9 + (Tiley<<1)+1)*128 + kz] = o;
			break;
		case 2:
			if (Tiley == 3) break;
			x0 = Inx*6, x1 = (Inx+1)*6;
			o00 = scale*(input[x0+1] + input[x0+2] + 4*input[x0+3] + 4*input[x0+4])+bias;
			o01 = scale*(input[x0+1] - input[x0+2] + 8*input[x0+3] - 8*input[x0+4] + input[x0+5])+bias;
			o10 = scale*(input[x1+1] + input[x1+2] + 4*input[x1+3] + 4*input[x1+4])+bias;
			o11 = scale*(input[x1+1] - input[x1+2] + 8*input[x1+3] - 8*input[x1+4] + input[x1+5])+bias;
			o = (0.25)*(relu(o00)+relu(o01)+relu(o10)+relu(o11));
			// o = fmaxf(fmaxf(relu(o00), relu(o01)), fmaxf(relu(o10), relu(o11)));
			pOutputs[(((Tilex<<1)+1+Inx/2)*9 + (Tiley<<1)+2)*128 + kz] = o;
			break;
	}

}

__global__ void kernel_128_winograd_AtIA_maxp(float *pInputs, float *pBiases, float *pScales, float *pOutputs) { 
	int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
	// int Tilex = threadIdx.x, Tiley = threadIdx.y, Iny = blockIdx.y, kz = blockIdx.z, Inx = blockIdx.x;

	int c_input = Inx*6 + Iny;

	__shared__ float bias, scale;
	 extern __shared__ float input[];

	input[c_input] = pInputs[c_input*16*128 + (Tilex*4+Tiley)*128 + kz];
	bias = pBiases[kz];
	scale = pScales[kz];
	__syncthreads();

	float tmp = 0;
	switch(Inx) {
		case 0:
			tmp = input[Iny] + input[6+Iny] + input[12+Iny] + input[18+Iny] + input[24+Iny];
			break;
		case 1:
			tmp = input[6+Iny] - input[12+Iny] + 2*input[18+Iny] - 2*input[24+Iny];
			break;
		case 2:
			tmp = input[6+Iny] + input[12+Iny] + 4*input[18+Iny] + 4*input[24+Iny];
			break;
		case 3:
			tmp = input[6+Iny] - input[12+Iny] + 8*input[18+Iny] - 8*input[24+Iny] + input[30+Iny];
			break;
	}
	__syncthreads();

	input[c_input] = tmp;
	__syncthreads();

	if (Inx >= 3 || Inx == 1 || (Tilex == 3 && Inx > 1)) return;
	int x0, x1;
	float o, o00, o01, o10, o11;
	switch(Iny) {
		case 0:
			x0 = Inx*6, x1 = (Inx+1)*6;
			o00 = input[x0]+input[x0+1]+input[x0+2]+input[x0+3]+input[x0+4];
			o01 = input[x0+1] - input[x0+2] + 2*input[x0+3] - 2*input[x0+4];
			o10 = input[x1]+input[x1+1]+input[x1+2]+input[x1+3]+input[x1+4];
			o11 = input[x1+1] - input[x1+2] + 2*input[x1+3] - 2*input[x1+4];
			// o = (0.25)*(relu(o00)+relu(o01)+relu(o10)+relu(o11));
			o = scale*fmaxf(fmaxf(o00, o01), fmaxf(o10, o11))+bias;
			pOutputs[(((Tilex<<1)+1+Inx/2)*9 + (Tiley<<1)+1)*128 + kz] = relu(o);
			break;
		case 2:
			if (Tiley == 3) break;
			x0 = Inx*6, x1 = (Inx+1)*6;
			o00 = input[x0+1] + input[x0+2] + 4*input[x0+3] + 4*input[x0+4];
			o01 = input[x0+1] - input[x0+2] + 8*input[x0+3] - 8*input[x0+4] + input[x0+5];
			o10 = input[x1+1] + input[x1+2] + 4*input[x1+3] + 4*input[x1+4];
			o11 = input[x1+1] - input[x1+2] + 8*input[x1+3] - 8*input[x1+4] + input[x1+5];
			// o = (0.25)*(relu(o00)+relu(o01)+relu(o10)+relu(o11));
			o = scale*fmaxf(fmaxf(o00, o01), fmaxf(o10, o11))+bias;
			pOutputs[(((Tilex<<1)+1+Inx/2)*9 + (Tiley<<1)+2)*128 + kz] = relu(o);
			break;
	}

}

__global__ void kernel_128_OuterProduct_128(float *A, float *B, float *C) {
	int Tile = blockIdx.x, Part = blockIdx.y, tX = threadIdx.x, tY = threadIdx.y;
	int c_input = tY*128 + tX, c_kernel = c_input, T_offset = (Tile<<11) + (Part<<10) + c_input, B_offset = (Tile<<14) + c_kernel;
	
	extern __shared__ float input[];
	float *kernel = input + 1024, *out = kernel + 8192;
	int B_stride[32] = {0, 128, 256, 384, 512, 640, 768, 896, 1024, 1152, 1280, 1408, 1536, 1664, 1792, 1920, 2048, 2176, 2304, 2432, 2560, 2688, 2816, 2944, 3072, 3200, 3328, 3456, 3584, 3712, 3840, 3968};//, 4096, 4224, 4352, 4480, 4608, 4736, 4864, 4992, 5120, 5248, 5376, 5504, 5632, 5760, 5888, 6016, 6144, 6272, 6400, 6528, 6656, 6784, 6912, 7040, 7168, 7296, 7424, 7552, 7680, 7808, 7936, 8064};
	out[c_input] = 0.0f;

	input[c_input] = A[T_offset];

	for (int k = 0; k < 4; k++) {
		int B_start = B_offset + (k<<12); // 32*64
		kernel[c_kernel] = B[B_start], kernel[c_kernel+1024] = B[B_start+1024];
		kernel[c_kernel+2048] = B[B_start+2048], kernel[c_kernel+3072] = B[B_start+3072];
		__syncthreads();

		float sum = 0;
		int y_tmp = (tY<<7)+(k<<5);
		for (int j = 0; j < 32; j++) {
			sum += input[y_tmp + j] * kernel[tX + B_stride[j]];
		}
		out[tY*128 + tX] += sum;
		__syncthreads();
	}

	C[T_offset] = out[c_input];
}

int kernel_128() {
	float *input_ = get_parameter(inputName128, 16*16*128);
	float *bias = get_parameter(biasName128, 128);
	float *input, *output, *l_weights, *l_bias, *pooling_output;
	uint64_t nT1 = 0, nT2 = 0, nT1_cudnn = 0, nT2_cudnn = 0;
	hipError_t s;
	hipdnnStatus_t status;

	float one = 1.0, zero = 0.0;

	/////////////////////////////////

	// My Kernel

	/////////////////////////////////


	/*  1. Data preparation  */
	float *t_input, *ip;
	//float *kernel = get_Winograd_Kernel128(weight_winograd_Name128, 128);
	float *kernel = get_parameter(weight_winograd_Name128, 36*128*128);
	float *l_bnBias, *l_bnScale, *bnBias, *bnScale;

	int nInput = 16*16*128, nOutput = 16*16*128, nWeights = 36*128*128, nBias = 128, nTransInput = 16*6*6*128, nInnerProd = 16*6*6*128, nPoolingOutput=9*9*128;
	hipMalloc((void **) &input, nInput<<3);
	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &pooling_output, nPoolingOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMalloc((void **) &t_input, nTransInput<<2);
	hipMalloc((void **) &ip, nInnerProd<<2);
	hipMemset((void *) input, 0, nInput<<3);
	hipMemset((void *) output, 0, nOutput<<2);
	hipMemset((void *) t_input, 0, nTransInput<<2);
	hipMemset((void *) l_weights, 0, nWeights<<2);
	hipMemset((void *) ip, 0, nInnerProd<<2);
	hipMemcpy(input, input_, nInput<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);
	
	bnBias = get_parameter(bnBias_winograd_Name128, 128);
	bnScale = get_parameter(bnScale_winograd_Name128, 128);
	hipMalloc((void **) &l_bnBias, nBias<<2);
	hipMalloc((void **) &l_bnScale, nBias<<2);
	hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
	float tmp_winograd[nOutput];
	float tmp_winograd_pooled[nPoolingOutput];

	hipdnnHandle_t win_handle;
	hipdnnTensorDescriptor_t winydesc, winpooldesc;
	status = hipdnnCreate(&win_handle);
	hipdnnPoolingDescriptor_t winpoolingDesc;
	status = hipdnnCreatePoolingDescriptor(&winpoolingDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed16\n");
	// HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING
	status = hipdnnSetPooling2dDescriptor(winpoolingDesc,
		// HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN, 2, 2, 1, 1, 2, 2);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed17\n");

	status = hipdnnCreateTensorDescriptor(&winpooldesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5.5\n");
	status = hipdnnSetTensor4dDescriptor(winpooldesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 9, 9);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5.51\n");

	status = hipdnnCreateTensorDescriptor(&winydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(winydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 16, 16);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	
	/*  2. Computing  */
	nT1 = getTimeMicroseconds64();

	kernel_128_winograd_BtdB <<<dim3(4, 4), dim3(128, 6), (6*6*128)<<2 >>> (input, t_input);
	kernel_128_OuterProduct_128<<<dim3(36, 2), dim3(128, 8), (8*128 + 64*128 + 8*128)<<2 >>> (t_input, l_weights, ip);
	// kernel_128_winograd_AtIA_maxp <<<dim3(4, 4, 128), dim3(6, 6), ((6*6)<<2)>>> (ip, l_bnBias, l_bnScale, pooling_output);
	// kernel_128_winograd_AtIA_avgp <<<dim3(4, 4, 128), dim3(6, 6), ((6*6)<<2)>>> (ip, l_bnBias, l_bnScale, pooling_output);
	// kernel_128_winograd_AtIA <<<dim3(4, 4, 128), dim3(6, 6), ((6*6)<<2)>>> (ip, l_bnBias, l_bnScale, output);
	kernel_128_winograd_AtIA_v2 <<<dim3(4, 4), dim3(128, 6), ((6*6*128)<<2)>>> (ip, l_bnBias, l_bnScale, output);

	// cudaCheckError();
	// status = hipdnnPoolingForward(win_handle, winpoolingDesc, &one,
	// 	winydesc, output, &zero,
	// 	winpooldesc, pooling_output);
	// if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed4\n");
	hipDeviceSynchronize();
	
	nT2 = getTimeMicroseconds64();
	printf("TotalTime = %d us\n", nT2-nT1); 


	/*  3. Copy back and free  */
	s = hipMemcpy(tmp_winograd, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	s = hipMemcpy(tmp_winograd_pooled, pooling_output, nPoolingOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));
	//cudaCheckError();
	// make_file("./tensors/winograd_out.bin", nOutput, tmp_winograd);
	// make_file("./tensors/winograd_out_pooled.bin", nPoolingOutput, tmp_winograd_pooled);

	hipFree(t_input);
	hipFree(output);
	hipFree(pooling_output);
	hipFree(l_weights);
	hipFree(l_bias);
	hipFree(ip);

	free(kernel);
	free(bnScale);
	free(bnBias);

	status = hipdnnDestroy(win_handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed16\n");


	/////////////////////////////////

	// cuDNN

	/////////////////////////////////

	/*  1. Data preparation  */
	kernel = get_parameter(weight_NCHW_Name128, 9*128*128);
	bnBias = get_parameter(bnBiasName128, 128);
	bnScale = get_parameter(bnScaleName128, 128);
	float* eMean = get_parameter(eMeanName128, 128);
	float* eVar = get_parameter(eVarName128, 128);
	float *l_eMean, *l_eVar;
	nInput = 16*16*128, nOutput = 14*14*128, nWeights = 3*3*128*128, nBias = 128, nPoolingOutput=7*7*128;

	hipMalloc((void **) &output, nOutput<<2);
	hipMalloc((void **) &pooling_output, nPoolingOutput<<2);
	hipMalloc((void **) &l_weights, nWeights<<2);
	hipMalloc((void **) &l_bias, nBias<<2);
	hipMemcpy(l_weights, kernel, nWeights<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bias, bias, nBias<<2, hipMemcpyHostToDevice);

	hipMalloc((void **) &l_eMean, nBias<<2);
	hipMalloc((void **) &l_eVar, nBias<<2);
	hipMemcpy(l_bnBias, bnBias, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_bnScale, bnScale, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eMean, eMean, nBias<<2, hipMemcpyHostToDevice);
	hipMemcpy(l_eVar, eVar, nBias<<2, hipMemcpyHostToDevice);

	hipMemset((void *) output, 0, nOutput<<2);

	float tmp_cudnn[nOutput];
	float tmp_pooled[nPoolingOutput];


	/*  2. cuDNN preparation  */
	int size;

	hipdnnHandle_t handle;
	status = hipdnnCreate(&handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed1\n");

	hipdnnTensorDescriptor_t xdesc, ydesc, bdesc, pooldesc;
	hipdnnFilterDescriptor_t wdesc; // HIPDNN_TENSOR_NHWC, HIPDNN_TENSOR_NCHW
	status = hipdnnCreateTensorDescriptor(&xdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed2\n");
	status = hipdnnSetTensor4dDescriptor(xdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 16, 16);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed3\n");
	status = hipdnnCreateTensorDescriptor(&ydesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed4\n");
	status = hipdnnSetTensor4dDescriptor(ydesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 14, 14);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5\n");
	status = hipdnnCreateTensorDescriptor(&pooldesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5.5\n");
	status = hipdnnSetTensor4dDescriptor(pooldesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 7, 7);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed5.51\n");
	status = hipdnnCreateFilterDescriptor(&wdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed6\n");
	status = hipdnnSetFilter4dDescriptor(wdesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, 128, 128, 3, 3);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed7\n");
	status = hipdnnCreateTensorDescriptor(&bdesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed8\n");
	status = hipdnnSetTensor4dDescriptor(bdesc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed9\n");
	hipdnnConvolutionDescriptor_t conv_desc;
	status = hipdnnCreateConvolutionDescriptor(&conv_desc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed10\n");
	status = hipdnnSetConvolution2dDescriptor(conv_desc, 0,0, 1,1,1,1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT); //HIPDNN_CONVOLUTION
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed11\n");

	hipdnnActivationDescriptor_t act_desc;
	status = hipdnnCreateActivationDescriptor(&act_desc);  
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed12\n");
	status = hipdnnSetActivationDescriptor(act_desc, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed13\n");
	hipdnnPoolingDescriptor_t poolingDesc;
	status = hipdnnCreatePoolingDescriptor(&poolingDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed16\n");
	// HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING
	status = hipdnnSetPooling2dDescriptor(poolingDesc,
		// HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING,
		HIPDNN_POOLING_MAX,
		HIPDNN_NOT_PROPAGATE_NAN, 2, 2, 0, 0, 2, 2);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed17\n");
	hipdnnTensorDescriptor_t bnScaleBiasMeanVarDesc;
	status = hipdnnCreateTensorDescriptor(&bnScaleBiasMeanVarDesc);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed14\n");
	status = hipdnnSetTensor4dDescriptor(bnScaleBiasMeanVarDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 128, 1, 1);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed15\n");


	hipdnnConvolutionFwdAlgo_t algo = (hipdnnConvolutionFwdAlgo_t)6;

	status = hipdnnGetConvolutionForwardWorkspaceSize(handle,
	   xdesc,
	   wdesc,
	   conv_desc,
	   ydesc,
	   algo,
	   (size_t *)&(size));

	float *extra;
	hipMalloc((void **) &extra, size);


	/*  3. Computing  */
	nT1_cudnn = getTimeMicroseconds64();

	status = hipdnnConvolutionForward(handle, &one,
		xdesc, input, wdesc, l_weights, 
		conv_desc, algo, 
		extra, size, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed1\n");

	status = hipdnnBatchNormalizationForwardInference(handle, HIPDNN_BATCHNORM_SPATIAL,
		&one, &zero, 
		ydesc, output, ydesc, output,
		bnScaleBiasMeanVarDesc, l_bnScale, l_bnBias, l_eMean, l_eVar, HIPDNN_BN_MIN_EPSILON);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed2\n");

	status = hipdnnActivationForward(handle, act_desc, &one,
		ydesc, output, &zero,
		ydesc, output);
	if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed3\n");

	// status = hipdnnPoolingForward(handle, poolingDesc, &one,
	// 	ydesc, output, &zero,
	// 	pooldesc, pooling_output);
	// if (status != HIPDNN_STATUS_SUCCESS) printf("Not Successed4\n");

	hipDeviceSynchronize();
	nT2_cudnn = getTimeMicroseconds64();
	printf("cuDNN TotalTime = %d us\n", nT2_cudnn-nT1_cudnn);


	/*  4. Copy back and free  */
	s = hipMemcpy(tmp_cudnn, output, nOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	s = hipMemcpy(tmp_pooled, pooling_output, nPoolingOutput<<2, hipMemcpyDeviceToHost);
	printf("%s\n", hipGetErrorName(s));

	// make_file("./tensors/pooled.bin", nPoolingOutput, tmp_pooled);
	// make_file("./tensors/cudnnout.bin", nOutput, tmp_cudnn);

	hipFree(extra);
	hipFree(input);
	hipFree(output);
	hipFree(pooling_output);
	hipFree(l_weights);
	hipFree(l_bias);

	hipFree(l_bnScale);
	hipFree(l_bnBias);
	hipFree(l_eMean);
	hipFree(l_eVar);

	free(bias);
	free(kernel);

	free(bnScale);
	free(bnBias);
	free(eMean);
	free(eVar);
	free(input_);
	status = hipdnnDestroy(handle);
	if (status != HIPDNN_STATUS_SUCCESS) printf("failed16\n");

	output_checker(tmp_winograd, tmp_cudnn, 14, 128, 1);
	// output_checker(tmp_winograd_pooled, tmp_pooled, 7, 128, 1);

	return ((nT2-nT1) << 16) | (nT2_cudnn-nT1_cudnn);
}